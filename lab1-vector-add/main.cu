#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <time.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;
    time_t t;


    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int n;
    if(argc == 1) {
        n = 10000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./vecadd               # Vector of size 10,000 is used"
           "\n    Usage: ./vecadd <m>           # Vector of size m is used"
           "\n");
        exit(0);
    }
    
    /* Intializes random number generator */
    srand((unsigned) time(&t));    
    

    float* A_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { A_h[i] = (rand()%100)/100.00; }

    float* B_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { B_h[i] = (rand()%100)/100.00; }

    float* C_h = (float*) malloc( sizeof(float)*n );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Vector size = %u\n", n);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    float* A_d;
    float* B_d;
    float* C_d;
    float bytes = sizeof(float) * n;
    
    hipMalloc(&A_d, bytes);
    hipMalloc(&B_d, bytes);
    hipMalloc(&C_d, bytes);
    
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    
    cuda_ret = hipMalloc((void**) &A_d, sizeof(float)*n);
    if(cuda_ret != hipSuccess) {
        printf("Testing hipMalloc A_d = Failed\n");
        exit(-1);
    }
    else {
        printf("Testing hipMalloc A_d = Passed\n");
    }
    
    cuda_ret = hipMalloc((void**) &B_d, sizeof(float)*n);
    if(cuda_ret != hipSuccess) {
        printf("Testing hipMalloc B_d = Failed\n");
        exit(-1);
    }
    else {
        printf("Testing hipMalloc B_d = Passed\n");
    }
    
    cuda_ret = hipMalloc((void**) &C_d, sizeof(float)*n);
    if(cuda_ret != hipSuccess) {
        printf("Testing hipMalloc C_d = Failed\n");
        exit(-1);
    }
    else {
        printf("Testing hipMalloc C_d = Passed\n");
    }

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    hipMemcpy(A_d, A_h, bytes, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, bytes, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    
    cuda_ret = hipMemcpy(A_d, A_h, sizeof(float)*n, hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) {
        printf("Testing hipMemcpy (A_d, A_h) = Failed\n");
        exit(-1);
    }
    else {
        printf("Testing hipMemcpy (A_d, A_h) = Passed\n");
    }
    
    cuda_ret = hipMemcpy(B_d, B_h, sizeof(float)*n, hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) {
        printf("Testing hipMemcpy (B_d, B_h) = Failed\n");
        exit(-1);
    }
    else {
        printf("Testing hipMemcpy (B_d, B_h) = Passed\n");
    }
    

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    float NUM_THREADS = 256.0;
    float NUM_BLOCKS = (float)ceil(n / NUM_THREADS);
    
    vecAddKernel<<<NUM_BLOCKS, NUM_THREADS>>>(A_d, B_d, C_d, n);

    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
	else {printf(" = Passed...");}
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    hipMemcpy(C_h, C_d, bytes, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    
    cuda_ret = hipMemcpy(C_h, C_d, bytes, hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) {
        printf("Testing hipMemcpy (C_h, C_d) = Failed\n");
        exit(-1);
    }
    else {
        printf("Testing hipMemcpy (C_h, C_d) = Passed\n");
    }

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, n);

    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    //INSERT CODE HERE
    cuda_ret = hipFree(A_d);
    if(cuda_ret != hipSuccess) {
        printf("Testing hipFree (A_d, A_d) = Failed\n");
        exit(-1);
    }
    else {
        printf("Testing hipFree (A_d, A_d) = Passed\n");
    }
    
    cuda_ret = hipFree(B_d);
    if(cuda_ret != hipSuccess) {
        printf("Testing hipFree (B_d, B_d) = Failed\n");
        exit(-1);
    }
    else {
        printf("Testing hipFree (B_d, B_d) = Passed\n");
    }
    
    cuda_ret = hipFree(C_d);
    if(cuda_ret != hipSuccess) {
        printf("Testing hipFree (C_h, C_d) = Failed\n");
        exit(-1);
    }
    else {
        printf("Testing hipFree (C_h, C_d) = Passed\n\n");
    }
    
    return 0;
}