
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {

    // Calculate global thread index based on the block and thread indices ----

    //INSERT KERNEL CODE HERE
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Use global index to determine which elements to read, add, and write ---

    //INSERT KERNEL CODE HERE
    if(tid < n) {
        C[tid] = A[tid] + B[tid];
    }
}